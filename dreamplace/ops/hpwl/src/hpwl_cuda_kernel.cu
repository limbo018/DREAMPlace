#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <float.h>
#include "hip/hip_runtime.h"
#include "utility/src/utils.cuh"

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
__global__ void computeHPWL(
        const T* x,
        const int* flat_netpin,
        const int* netpin_start,
        const unsigned char* net_mask,
        int num_nets,
        T* partial_hpwl
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_nets)
    {
        T max_x = -FLT_MAX;
        T min_x = FLT_MAX;

        if (net_mask[i])
        {
            for (int j = netpin_start[i]; j < netpin_start[i+1]; ++j)
            {
                min_x = min(min_x, x[flat_netpin[j]]);
                max_x = max(max_x, x[flat_netpin[j]]);
            }
            partial_hpwl[i] = max_x-min_x;
        }
        else
        {
            partial_hpwl[i] = 0;
        }
    }
}

template <typename T>
int computeHPWLCudaLauncher(
        const T* x, const T* y,
        const int* flat_netpin,
        const int* netpin_start,
        const unsigned char* net_mask,
        int num_nets,
        T* partial_hpwl
        )
{
    const int thread_count = 512;
    const int block_count_nets = (num_nets + thread_count - 1) / thread_count;

    hipError_t status;
    hipStream_t stream_y;
    status = hipStreamCreate(&stream_y);
    if (status != hipSuccess)
    {
        printf("hipStreamCreate failed for stream_y\n");
        fflush(stdout);
        return 1;
    }

    computeHPWL<<<block_count_nets, thread_count>>>(
            x,
            flat_netpin,
            netpin_start,
            net_mask,
            num_nets,
            partial_hpwl
            );

    computeHPWL<<<block_count_nets, thread_count, 0, stream_y>>>(
            y,
            flat_netpin,
            netpin_start,
            net_mask,
            num_nets,
            partial_hpwl+num_nets
            );

    /* destroy stream */
    status = hipStreamDestroy(stream_y);
    if (status != hipSuccess)
    {
        printf("stream_y destroy failed\n");
        fflush(stdout);
        return 1;
    }

    //printArray(partial_hpwl, num_nets, "partial_hpwl");

    // I move out the summation to use ATen
    // significant speedup is observed
    //sumArray<<<1, 1>>>(partial_hpwl, num_nets, hpwl);

    return 0;
}

// manually instantiate the template function
#define REGISTER_KERNEL_LAUNCHER(type) \
    template int computeHPWLCudaLauncher<type>(\
        const type* x, const type* y, \
        const int* flat_netpin, \
        const int* netpin_start, \
        const unsigned char* net_mask, \
        int num_nets, \
        type* partial_hpwl \
        ); 

REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
