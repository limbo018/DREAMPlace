#include "hip/hip_runtime.h"
/**
 * @file   integrate_net_weights_cuda_kernel.cu
 * @author Yibo Lin
 * @date   Jul 2019
 */

#include <cfloat>
#include <stdio.h>
#include "assert.h"
#include "hip/hip_runtime.h"
#include "weighted_average_wirelength/src/functional_cuda.h"
#include "utility/src/utils.cuh"

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
__global__ void integrateNetWeights(
        const int* pin2net_map, 
        const unsigned char* net_mask, 
        const T* net_weights, 
        T* grad_x_tensor, T* grad_y_tensor, 
        int num_pins
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_pins)
    {
        int net_id = pin2net_map[i]; 
        T weight = net_weights[net_id]; 
        if (net_id >= 0 && net_mask[net_id])
        {
            grad_x_tensor[i] *= weight; 
            grad_y_tensor[i] *= weight; 
        }
    }
}

template <typename T>
void integrateNetWeightsCudaLauncher(
        const int* pin2net_map, 
        const unsigned char* net_mask, 
        const T* net_weights, 
        T* grad_x_tensor, T* grad_y_tensor, 
        int num_pins
        )
{
    integrateNetWeights<<<ceilDiv(num_pins, 256), 256>>>(pin2net_map, net_mask, net_weights, grad_x_tensor, grad_y_tensor, num_pins); 
}

template <typename T>
__global__ void integrateNetWeightsforWL(
        const int* pin2net_map, 
        const unsigned char* net_mask, 
        const T* net_weights, 
        T* partial_wl, ///< x,y interleave
        int num_pins
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_pins)
    {
        int net_id = pin2net_map[i]; 
        T weight = net_weights[net_id]; 
        if (net_id >= 0 && net_mask[net_id])
        {
            partial_wl[2*i] *= weight; 
            partial_wl[2*i+1] *= weight; 
        }
    }
}

template <typename T>
void integrateNetWeightsforWLCudaLauncher(
        const int* pin2net_map, 
        const unsigned char* net_mask, 
        const T* net_weights, 
        T* partial_wl, 
        int num_pins
        )
{
    integrateNetWeightsforWL<<<ceilDiv(num_pins, 256), 256>>>(pin2net_map, net_mask, net_weights, partial_wl, num_pins); 
}

#define REGISTER_KERNEL_LAUNCHER(T) \
    template void integrateNetWeightsCudaLauncher<T>(\
        const int* pin2net_map, \
        const unsigned char* net_mask, \
        const T* net_weights, \
        T* grad_x_tensor, T* grad_y_tensor, \
        int num_pins \
        ); \
    template void integrateNetWeightsforWLCudaLauncher<T>(\
        const int* pin2net_map, \
        const unsigned char* net_mask, \
        const T* net_weights, \
        T* partial_wl, \
        int num_pins \
        );

REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
