#include "hip/hip_runtime.h"
#include <cfloat>
#include <stdio.h>
#include "assert.h"
#include "hip/hip_runtime.h"
#include "utility/src/utils.cuh"

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
__global__ void computeLogSumExpWirelength(
        const T *x, const T *y,
        const int *flat_netpin,
        const int *netpin_start,
        const unsigned char *net_mask,
        int num_nets,
        const T* gamma, 
        const T *inv_gamma,
        T *partial_wl,
        T *grad_intermediate_x, T *grad_intermediate_y
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int ii = i >> 1;
    if (ii < num_nets && net_mask[ii])
    {
        const T *values;
        T *grads;
        if (i & 1)
        {
            values = y;
            grads = grad_intermediate_y;
        }
        else
        {
            values = x;
            grads = grad_intermediate_x;
        }

        // int degree = netpin_start[ii+1]-netpin_start[ii];
        T x_max = -FLT_MAX;
        T x_min = FLT_MAX;
        for (int j = netpin_start[ii]; j < netpin_start[ii + 1]; ++j)
        {
            T xx = values[flat_netpin[j]];
            x_max = max(xx, x_max);
            x_min = min(xx, x_min);
        }

        T exp_x_sum = 0;
        T exp_nx_sum = 0;
        for (int j = netpin_start[ii]; j < netpin_start[ii + 1]; ++j)
        {
            T xx = values[flat_netpin[j]];
            T exp_x = exp((xx - x_max) * (*inv_gamma));
            T exp_nx = exp((x_min - xx) * (*inv_gamma));

            exp_x_sum += exp_x;
            exp_nx_sum += exp_nx;
        }

        partial_wl[i] = (log(exp_x_sum) + log(exp_nx_sum)) * (*gamma) + x_max - x_min;

        T reciprocal_exp_x_sum = 1.0 / exp_x_sum; 
        T reciprocal_exp_nx_sum = 1.0 / exp_nx_sum; 
        for (int j = netpin_start[ii]; j < netpin_start[ii+1]; ++j)
        {
            int jj = flat_netpin[j];
            T xx = values[jj];
            T exp_x = exp((xx - x_max) * (*inv_gamma));
            T exp_nx = exp((x_min - xx) * (*inv_gamma));
            grads[jj] = (exp_x*reciprocal_exp_x_sum - exp_nx*reciprocal_exp_nx_sum); 
        }
    }
}

template <typename T>
int computeLogSumExpWirelengthCudaLauncher(
        const T* x, const T* y, 
        const int* flat_netpin, 
        const int* netpin_start, 
        const unsigned char* net_mask, 
        int num_nets,
        const T* gamma, 
        const T* inv_gamma, 
        T* partial_wl,
        T* grad_intermediate_x, T* grad_intermediate_y
        )
{
    int thread_count = 64;
    int block_count = (num_nets * 2 + thread_count - 1) / thread_count; // separate x and y

    computeLogSumExpWirelength<<<block_count, thread_count>>>(
        x, y,
        flat_netpin,
        netpin_start,
        net_mask,
        num_nets,
        gamma, 
        inv_gamma,
        partial_wl,
        grad_intermediate_x, grad_intermediate_y
        );

    return 0;
}


#define REGISTER_KERNEL_LAUNCHER(T) \
    template int computeLogSumExpWirelengthCudaLauncher<T>(\
                const T* x, const T* y,       \
                const int* flat_netpin,       \
                const int* netpin_start,       \
                const unsigned char* net_mask,       \
                int num_nets,      \
                const T* gamma,       \
                const T* inv_gamma,       \
                T* partial_wl,      \
                T* grad_intermediate_x, T* grad_intermediate_y      \
            ); 

REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
