#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <float.h>
#include "hip/hip_runtime.h"
#include "utility/src/utils.cuh"

DREAMPLACE_BEGIN_NAMESPACE


template <typename T>
__global__ void computeFlip(
        const T* x, 
        const int M, 
        const int N, 
        T* y
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M*N; i += blockDim.x * gridDim.x) 
    {
        int ii = i%N; 
        y[i] = x[i+N-ii*2-1];
    }
}

template <typename T>
void computeFlipCudaLauncher(
        const T* x, 
        const int M, 
        const int N, 
        T* y
        )
{
    computeFlip<<<32, 1024>>>(
            x, 
            M, 
            N, 
            y
            );
}

template <typename T>
__global__ void computeFlipAndShift(
        const T* x, 
        const int M, 
        const int N, 
        T* y
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M*N; i += blockDim.x * gridDim.x) 
    {
        int ii = i%N; 
        y[i] = (ii)? x[i+N-ii*2] : 0;
    }
}

template <typename T>
void computeFlipAndShiftCudaLauncher(
        const T* x, 
        const int M, 
        const int N, 
        T* y
        )
{
    computeFlipAndShift<<<32, 1024>>>(
            x, 
            M, 
            N, 
            y
            );
}

template <typename T>
__global__ void negateOddEntries(
        T* x, 
        const int M, 
        const int N
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M*(N>>1); i += blockDim.x * gridDim.x) 
    {
        x[i*2+1] = -x[i*2+1]; 
    }
}

template <typename T>
void negateOddEntriesCudaLauncher(
        T* x, 
        const int M, 
        const int N
        )
{
    negateOddEntries<<<32, 1024>>>(
            x, 
            M, 
            N
            );
}

#define REGISTER_FLIP_KERNEL_LAUNCHER(type) \
    template void computeFlipCudaLauncher<type>(\
        const type* x, \
        const int M, \
        const int N, \
        type* y \
        ); 

REGISTER_FLIP_KERNEL_LAUNCHER(float);
REGISTER_FLIP_KERNEL_LAUNCHER(double);

#define REGISTER_FLIPANDSHIFT_KERNEL_LAUNCHER(type) \
    template void computeFlipAndShiftCudaLauncher<type>(\
        const type* x, \
        const int M, \
        const int N, \
        type* y \
        ); 

REGISTER_FLIPANDSHIFT_KERNEL_LAUNCHER(float);
REGISTER_FLIPANDSHIFT_KERNEL_LAUNCHER(double);

#define REGISTER_NEGATE_KERNEL_LAUNCHER(type) \
    template void negateOddEntriesCudaLauncher<type>(\
        type* x, \
        const int M, \
        const int N \
        ); 

REGISTER_NEGATE_KERNEL_LAUNCHER(float);
REGISTER_NEGATE_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
