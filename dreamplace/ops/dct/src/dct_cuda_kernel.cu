#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <float.h>
#include "hip/hip_runtime.h"
#include "utility/src/utils.cuh"

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
__global__ void computeMulExpk(
        const T* x,
        const T* expk,
        const int M,
        const int N,
        T* z
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < M*N)
    {
        int row = i/N; // row
        int col = i-row*N; // column
        int col_2x = (col<<1);
        int fft_onesided_size = (N>>1)+1;
        int fft_onesided_size_2x = fft_onesided_size<<1;

        if (col_2x <= N)
        {
            int j = row*fft_onesided_size_2x + col_2x;
            //printf("x[%d]*expk[%d] + x[%d]*expk[%d] = z[%d]\n", j, col_2x, j+1, col_2x+1, i);
            z[i] = x[j]*expk[col_2x] + x[j+1]*expk[col_2x+1];
        }
        else
        {
            int j = row*fft_onesided_size_2x + (N<<1) - col_2x;
            //printf("x[%d]*expk[%d] + x[%d]*expk[%d] = z[%d]\n", j, col_2x, j+1, col_2x+1, i);
            z[i] = x[j]*expk[col_2x] - x[j+1]*expk[col_2x+1];
        }
    }
}

template <typename T>
void computeMulExpkCudaLauncher(
        const T* x,
        const T* expk,
        const int M,
        const int N,
        T* z
        )
{
    const int thread_count = 1024;
    const int block_count = (M * N - 1 + thread_count) / thread_count;

    computeMulExpk<<<block_count, thread_count>>>(
            x,
            expk,
            M,
            N,
            z
            );
}

template <typename T>
__global__ void computeReorder(
        const T* x,
        const int M,
        const int N,
        T* y
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < M*N)
    {
        int ii = i%N;

        if (ii < (N>>1))
        {
            // i*2
            //printf("x[%d] = y[%d]\n", i+ii, i);
            y[i] = x[i+ii];
        }
        else
        {
            // (N-i)*2-1
            //printf("x[%d] = y[%d]\n", i+N*2-ii*3-1, i);
            y[i] = x[i+N*2-ii*3-1];
        }
    }
}

template <typename T>
void computeReorderCudaLauncher(
        const T* x,
        const int M,
        const int N,
        T* y
        )
{
    const int thread_count = 1024;
    const int block_count = (M * N - 1 + thread_count) / thread_count;

    computeReorder<<<block_count, thread_count>>>(
            x,
            M,
            N,
            y
            );
}

template <typename T>
__global__ void computeVk(
        const T* x,
        const T* expk,
        const int M,
        const int N,
        T* v
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < M*(N/2+1))
    {
        int ncol = N/2+1;
        int row = i/ncol; // row
        int col = i-row*ncol; // column
        int col_2x = (col<<1);

        // real
        T real = x[row*N+col];
        T imag = (col == 0)? 0 : -x[row*N+N-col];

        v[2*i] = real*expk[col_2x] - imag*expk[col_2x+1];
        // imag, x[N-i]
        v[2*i+1] = real*expk[col_2x+1] + imag*expk[col_2x];
    }

}

template <typename T>
void computeVkCudaLauncher(
        const T* x,
        const T* expk,
        const int M,
        const int N,
        T* v
        )
{
    const int thread_count = 512;
    const int block_count = (M*(N/2+1) - 1 + thread_count) / thread_count;

    computeVk<<<block_count, thread_count>>>(
            x,
            expk,
            M,
            N,
            v
            );
}


template <typename T>
__global__ void computeReorderReverse(
        const T* y,
        const int M,
        const int N,
        T* z
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < M*N)
    {
        int row = i/N; // row
        int col = i-row*N; // column

        //printf("z[%d] = y[%d]\n", i, (col&1)? (i-col*3/2+N-1) : (i-col/2));
        //z[i] = (col&1)? y[(i-col*3/2+N-1)] : y[(i-col/2)];
        // according to the paper, it should be N - (col+1)/2 for col is odd
        // but it seems previous implementation accidentally matches this as well
        z[i] = (col&1)? y[(i-col) + N - (col+1)/2] : y[(i-col/2)];
    }
}

template <typename T>
void computeReorderReverseCudaLauncher(
        const T* y,
        const int M,
        const int N,
        T* z
        )
{
    const int thread_count = 512;
    const int block_count = (M * N - 1 + thread_count) / thread_count;

    computeReorderReverse<<<block_count, thread_count>>>(
            y,
            M,
            N,
            z
            );
}

template <typename T>
__global__ void addX0AndScale(
        const T* x,
        const int M,
        const int N,
        T* y
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M*N; i += blockDim.x * gridDim.x)
    {
        int i0 = int(i/N)*N;
        y[i] = (y[i]+x[i0])*0.5;
    }
}

template <typename T>
void addX0AndScaleCudaLauncher(
        const T* x,
        const int M,
        const int N,
        T* y
        )
{
    addX0AndScale<<<32, 1024>>>(
            x,
            M,
            N,
            y
            );
}

/// extends from addX0AndScale to merge scaling
template <typename T>
__global__ void addX0AndScaleN(
        const T* x,
        const int M,
        const int N,
        T* y
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M*N; i += blockDim.x * gridDim.x)
    {
        int i0 = int(i/N)*N;
        // this is to match python implementation
        // normal way should be multiply by 0.25*N
        y[i] = y[i]*0.25*N+x[i0]*0.5;
    }
}

template <typename T>
void addX0AndScaleNCudaLauncher(
        const T* x,
        const int M,
        const int N,
        T* y
        )
{
    addX0AndScaleN<<<32, 1024>>>(
            x,
            M,
            N,
            y
            );
}

template <typename T>
__global__ void computePad(
        const T* x, // M*N
        const int M,
        const int N,
        T* z // M*2N
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M*N; i += blockDim.x * gridDim.x)
    {
        int row = i/N; // row
        int col = i-row*N; // column
        int j = row*(N<<1) + col;
        z[j] = x[i];
    }
}

template <typename T>
void computePadCudaLauncher(
        const T* x, // M*N
        const int M,
        const int N,
        T* z // M*2N
        )
{
    computePad<<<32, 1024>>>(
            x,
            M,
            N,
            z
            );
}

template <typename T>
__global__ void computeMulExpk_2N(
        const T* x, // M*(N+1)*2
        const T* expk,
        const int M,
        const int N,
        T* z // M*N
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M*N; i += blockDim.x * gridDim.x)
    {
        int row = i/N; // row
        int col = i-row*N; // column
        int col_2x = (col<<1);
        int j = row*((N+1)<<1) + col_2x;
        z[i] = x[j]*expk[col_2x] + x[j+1]*expk[col_2x+1];
    }
}

template <typename T>
void computeMulExpk_2N_CudaLauncher(
        const T* x, // M*(N+1)*2
        const T* expk,
        const int M,
        const int N,
        T* z // M*N
        )
{
    computeMulExpk_2N<<<32, 1024>>>(
            x,
            expk,
            M,
            N,
            z
            );
}

template <typename T>
__global__ void computeMulExpkAndPad_2N(
        const T* x, // M*N
        const T* expk,
        const int M,
        const int N,
        T* z // M*2N*2
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M*N; i += blockDim.x * gridDim.x)
    {
        int row = i/N; // row
        int col = i-row*N; // column
        int col_2x = (col<<1);
        int j = row*(N<<2) + col_2x;
        z[j] = x[i]*expk[col_2x];
        z[j+1] = x[i]*expk[col_2x+1];
    }
}

template <typename T>
void computeMulExpkAndPad_2N_CudaLauncher(
        const T* x, // M*N
        const T* expk,
        const int M,
        const int N,
        T* z // M*2N*2
        )
{
    computeMulExpkAndPad_2N<<<32, 1024>>>(
            x,
            expk,
            M,
            N,
            z
            );
}

/// remove last N entries in each column
template <typename T>
__global__ void computeTruncation(
        const T* x, // M*2N
        const int M,
        const int N,
        T* z // M*N
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M*N; i += blockDim.x * gridDim.x)
    {
        int row = i/N; // row
        int col = i-row*N; // column
        int j = row*(N<<1) + col;
        z[i] = x[j];
    }
}

template <typename T>
void computeTruncationCudaLauncher(
        const T* x, // M*2N
        const int M,
        const int N,
        T* z // M*N
        )
{
    computeTruncation<<<32, 1024>>>(
            x,
            M,
            N,
            z
            );
}

// manually instantiate the template function
#define REGISTER_MULPEXPK_KERNEL_LAUNCHER(type) \
    template void computeMulExpkCudaLauncher<type>(\
        const type* x, \
        const type* expk, \
        const int M, \
        const int N, \
        type* z \
        ); 
    
REGISTER_MULPEXPK_KERNEL_LAUNCHER(float);
REGISTER_MULPEXPK_KERNEL_LAUNCHER(double);

#define REGISTER_REORDER_KERNEL_LAUNCHER(type) \
    template void computeReorderCudaLauncher<type>(\
        const type* x, \
        const int M, \
        const int N, \
        type* y \
        ); 
    

REGISTER_REORDER_KERNEL_LAUNCHER(float);
REGISTER_REORDER_KERNEL_LAUNCHER(double);

#define REGISTER_VK_KERNEL_LAUNCHER(type) \
    template void computeVkCudaLauncher<type>(\
        const type* x, \
        const type* expk, \
        const int M, \
        const int N, \
        type* v \
        ); 

REGISTER_VK_KERNEL_LAUNCHER(float);
REGISTER_VK_KERNEL_LAUNCHER(double);

#define REGISTER_REORDERREVERSE_KERNEL_LAUNCHER(type) \
    template void computeReorderReverseCudaLauncher<type>(\
        const type* y, \
        const int M, \
        const int N, \
        type* z \
        ); 

REGISTER_REORDERREVERSE_KERNEL_LAUNCHER(float);
REGISTER_REORDERREVERSE_KERNEL_LAUNCHER(double);

#define REGISTER_ADDX0ANDSCALE_KERNEL_LAUNCHER(type) \
    template void addX0AndScaleCudaLauncher<type>(\
        const type* x, \
        const int M, \
        const int N, \
        type* y \
        ); 

REGISTER_ADDX0ANDSCALE_KERNEL_LAUNCHER(float);
REGISTER_ADDX0ANDSCALE_KERNEL_LAUNCHER(double);

#define REGISTER_ADDX0ANDSCALEN_KERNEL_LAUNCHER(type) \
    template void addX0AndScaleNCudaLauncher<type>(\
        const type* x, \
        const int M, \
        const int N, \
        type* y \
        ); 

REGISTER_ADDX0ANDSCALEN_KERNEL_LAUNCHER(float);
REGISTER_ADDX0ANDSCALEN_KERNEL_LAUNCHER(double);

#define REGISTER_COMPUTEPAD_KERNEL_LAUNCHER(type) \
    template void computePadCudaLauncher<type>(\
            const type* x, \
            const int M, \
            const int N, \
            type* z \
        ); 

REGISTER_COMPUTEPAD_KERNEL_LAUNCHER(float);
REGISTER_COMPUTEPAD_KERNEL_LAUNCHER(double);

#define REGISTER_COMPUTEMULEXPK_2N_KERNEL_LAUNCHER(type) \
    template void computeMulExpk_2N_CudaLauncher<type>(\
            const type* x, \
            const type* expk, \
            const int M, \
            const int N, \
            type* z \
        ); 

REGISTER_COMPUTEMULEXPK_2N_KERNEL_LAUNCHER(float);
REGISTER_COMPUTEMULEXPK_2N_KERNEL_LAUNCHER(double);

#define REGISTER_COMPUTEMULEXPKANDPAD_2N_KERNEL_LAUNCHER(type) \
    template void computeMulExpkAndPad_2N_CudaLauncher<type>(\
            const type* x, \
            const type* expk, \
            const int M, \
            const int N, \
            type* z \
        ); 

REGISTER_COMPUTEMULEXPKANDPAD_2N_KERNEL_LAUNCHER(float);
REGISTER_COMPUTEMULEXPKANDPAD_2N_KERNEL_LAUNCHER(double);

#define REGISTER_COMPUTETRUNCATION_KERNEL_LAUNCHER(type) \
    template void computeTruncationCudaLauncher<type>(\
            const type* x, \
            const int M, \
            const int N, \
            type* z \
        ); 

REGISTER_COMPUTETRUNCATION_KERNEL_LAUNCHER(float);
REGISTER_COMPUTETRUNCATION_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
