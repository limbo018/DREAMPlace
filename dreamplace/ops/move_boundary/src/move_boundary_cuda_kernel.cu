#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <float.h>
#include "hip/hip_runtime.h"
#include "utility/src/utils.cuh"

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
__global__ void computeMoveBoundary(
        T* x_tensor,
        const T* node_size_x_tensor,
        const T xl, const T xh,
        const int num_nodes,
        const int num_movable_nodes,
        const int num_filler_nodes
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_movable_nodes || (i >= num_nodes-num_filler_nodes && i < num_nodes))
    {
        x_tensor[i] = min(xh-node_size_x_tensor[i], max(xl, x_tensor[i]));
    }
}

template <typename T>
int computeMoveBoundaryMapCudaLauncher(
        T* x_tensor, T* y_tensor,
        const T* node_size_x_tensor, const T* node_size_y_tensor,
        const T xl, const T yl, const T xh, const T yh,
        const int num_nodes,
        const int num_movable_nodes,
        const int num_filler_nodes
        )
{
    hipError_t status;
    hipStream_t stream_y;
    status = hipStreamCreate(&stream_y);
    if (status != hipSuccess)
    {
        printf("hipStreamCreate failed for stream_y\n");
        fflush(stdout);
        return 1;
    }

    int thread_count = 512;
    int block_count = (num_nodes - 1 + thread_count) / thread_count;
    computeMoveBoundary<<<block_count, thread_count>>>(
            x_tensor,
            node_size_x_tensor,
            xl, xh,
            num_nodes,
            num_movable_nodes,
            num_filler_nodes
            );

    computeMoveBoundary<<<block_count, thread_count, 0, stream_y>>>(
            y_tensor,
            node_size_y_tensor,
            yl, yh,
            num_nodes,
            num_movable_nodes,
            num_filler_nodes
            );

    /* destroy stream */
    status = hipStreamDestroy(stream_y);
    if (status != hipSuccess)
    {
        printf("stream_y destroy failed\n");
        fflush(stdout);
        return 1;
    }
    return 0;
}

#define REGISTER_KERNEL_LAUNCHER(T) \
  template int computeMoveBoundaryMapCudaLauncher<T>(\
            T* x_tensor, T* y_tensor, \
            const T* node_size_x_tensor, const T* node_size_y_tensor, \
            const T xl, const T yl, const T xh, const T yh, \
            const int num_nodes, \
            const int num_movable_nodes, \
            const int num_filler_nodes \
            );
    
REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
