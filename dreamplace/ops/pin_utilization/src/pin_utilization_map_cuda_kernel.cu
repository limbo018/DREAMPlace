#include "hip/hip_runtime.h"
/**
 * @file   pin_utilization_map_cuda_kernel.cu
 * @author Zixuan Jiang, Jiaqi Gu, Yibo Lin
 * @date   Dec 2019
 * @brief  Compute the RUDY/RISA map for routing demand. 
 *         A routing/pin utilization estimator based on the following two papers
 *         "Fast and Accurate Routing Demand Estimation for efficient Routability-driven Placement", by Peter Spindler, DATE'07
 *         "RISA: Accurate and Efficient Placement Routability Modeling", by Chih-liang Eric Cheng, ICCAD'94
 */

#include "utility/src/utils.cuh"

DREAMPLACE_BEGIN_NAMESPACE

// fill the demand map net by net
template <typename T, typename AtomicOp>
__global__ void pinDemandMap(const T *node_x, const T *node_y,
                          const T *node_size_x, const T *node_size_y,
                          const T *half_node_size_stretch_x, const T *half_node_size_stretch_y,
                          const T *pin_weights,
                          T xl, T yl, T xh, T yh,
                          T bin_size_x, T bin_size_y,
                          int num_bins_x, int num_bins_y,
                          int num_nodes, AtomicOp atomic_add_op,
                          typename AtomicOp::type *buf_map
                          )
{
    const int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < num_nodes)
    {
        const T node_center_x = node_x[i] + node_size_x[i]/2; 
        const T node_center_y = node_y[i] + node_size_y[i]/2; 

        const T x_min = node_center_x - half_node_size_stretch_x[i];
        const T x_max = node_center_x + half_node_size_stretch_x[i];
        int bin_index_xl = int((x_min - xl) / bin_size_x);
        int bin_index_xh = int((x_max - xl) / bin_size_x) + 1;
        bin_index_xl = DREAMPLACE_STD_NAMESPACE::max(bin_index_xl, 0);
        bin_index_xh = DREAMPLACE_STD_NAMESPACE::min(bin_index_xh, num_bins_x);

        const T y_min = node_center_y - half_node_size_stretch_y[i];
        const T y_max = node_center_y + half_node_size_stretch_y[i];
        int bin_index_yl = int((y_min - yl) / bin_size_y);
        int bin_index_yh = int((y_max - yl) / bin_size_y) + 1;
        bin_index_yl = DREAMPLACE_STD_NAMESPACE::max(bin_index_yl, 0);
        bin_index_yh = DREAMPLACE_STD_NAMESPACE::min(bin_index_yh, num_bins_y);

        T density = pin_weights[i] / (half_node_size_stretch_x[i] * half_node_size_stretch_y[i] * 4);
        for (int x = bin_index_xl; x < bin_index_xh; ++x)
        {
            for (int y = bin_index_yl; y < bin_index_yh; ++y)
            {
                T bin_xl = xl + x * bin_size_x; 
                T bin_yl = yl + y * bin_size_y; 
                T bin_xh = bin_xl + bin_size_x; 
                T bin_yh = bin_yl + bin_size_y; 
                T overlap = DREAMPLACE_STD_NAMESPACE::max(DREAMPLACE_STD_NAMESPACE::min(x_max, bin_xh) - DREAMPLACE_STD_NAMESPACE::max(x_min, bin_xl), (T)0) *
                            DREAMPLACE_STD_NAMESPACE::max(DREAMPLACE_STD_NAMESPACE::min(y_max, bin_yh) - DREAMPLACE_STD_NAMESPACE::max(y_min, bin_yl), (T)0);
                int index = x * num_bins_y + y;
                atomic_add_op(&buf_map[index], overlap * density);
            }
        }
    }
}

// fill the demand map net by net
template <typename T>
int pinDemandMapCudaLauncher(const T *node_x, const T *node_y,
                          const T *node_size_x, const T *node_size_y,
                          const T *half_node_size_stretch_x, const T *half_node_size_stretch_y,
                          const T *pin_weights,
                          T xl, T yl, T xh, T yh,
                          int num_bins_x, int num_bins_y,
                          int num_nodes,
                          bool deterministic_flag, 
                          T *pin_utilization_map
                          )
{
  T bin_size_x = (xh - xl) / num_bins_x; 
  T bin_size_y = (yh - yl) / num_bins_y; 

  if (deterministic_flag)  // deterministic implementation using unsigned long
                           // as fixed point number
  {
    // total die area
    double diearea = (xh - xl) * (yh - yl);
    int integer_bits = max((int)ceil(log2(diearea)) + 1, 32);
    int fraction_bits = max(64 - integer_bits, 0);
    unsigned long long int scale_factor = (1UL << fraction_bits);
    int num_bins = num_bins_x * num_bins_y;
    unsigned long long int *buf_map = NULL;
    allocateCUDA(buf_map, num_bins, unsigned long long int);

    AtomicAddCUDA<unsigned long long int> atomic_add_op(scale_factor);

    int thread_count = 512;
    int block_count = ceilDiv(num_bins, thread_count);
    copyScaleArray<<<block_count, thread_count>>>(
        buf_map, pin_utilization_map, scale_factor, num_bins);

    block_count = ceilDiv(num_nodes, thread_count);
    pinDemandMap<<<block_count, thread_count>>>(
            node_x, node_y,
            node_size_x, node_size_y,
            half_node_size_stretch_x, half_node_size_stretch_y,
            pin_weights,
            xl, yl, xh, yh,
            bin_size_x, bin_size_y,
            num_bins_x, num_bins_y,
            num_nodes,
            atomic_add_op, 
            buf_map
        );

    block_count = ceilDiv(num_bins, thread_count);
    copyScaleArray<<<block_count, thread_count>>>(
        pin_utilization_map, buf_map, T(1.0 / scale_factor), num_bins);

    destroyCUDA(buf_map);
  } else {
    AtomicAddCUDA<T> atomic_add_op;
    int thread_count = 512;
    int block_count = ceilDiv(num_nodes, thread_count);
    pinDemandMap<<<block_count, thread_count>>>(
            node_x, node_y,
            node_size_x, node_size_y,
            half_node_size_stretch_x, half_node_size_stretch_y,
            pin_weights,
            xl, yl, xh, yh,
            bin_size_x, bin_size_y,
            num_bins_x, num_bins_y,
            num_nodes,
            atomic_add_op,
            pin_utilization_map
        );
  }

    return 0;
}

#define REGISTER_KERNEL_LAUNCHER(T)                                                                       \
    template int pinDemandMapCudaLauncher<T>(const T *node_x, const T *node_y, \
            const T *node_size_x, const T *node_size_y, \
            const T *half_node_size_stretch_x, const T *half_node_size_stretch_y, \
            const T *pin_weights, \
            T xl, T yl, T xh, T yh, \
            int num_bins_x, int num_bins_y, \
            int num_nodes, \
            bool deterministic_flag, \
            T *pin_utilization_map \
            );

REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
