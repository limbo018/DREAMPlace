#include "hip/hip_runtime.h"
#include "utility/src/utils.cuh"

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
__global__ void updatePinOffset(
    const T* node_size_x, const T* node_size_y, 
    const int *flat_node2pin_start_map,
    const int *flat_node2pin_map,
    const T *node_ratios,
    const int num_nodes,
    T *pin_offset_x, T *pin_offset_y
    )
{
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < num_nodes)
    {
        T ratio = (node_ratios[i] - 1) / 2;
        T sx = node_size_x[i]; 
        T sy = node_size_y[i]; 

        int start = flat_node2pin_start_map[i];
        int end = flat_node2pin_start_map[i + 1];
        for (int j = start; j < end; ++j)
        {
            int pin_id = flat_node2pin_map[j];
            pin_offset_x[pin_id] += ratio * sx;
            pin_offset_y[pin_id] += ratio * sy;
        }
    }
}

template <typename T>
void updatePinOffsetCudaLauncher(
    const T* node_size_x, const T* node_size_y, 
    const int *flat_node2pin_start_map,
    const int *flat_node2pin_map,
    const T *node_ratios,
    const int num_nodes,
    T *pin_offset_x, T *pin_offset_y
    )
{
    int thread_count = 512;
    int block_count = ceilDiv(num_nodes, thread_count);
    updatePinOffset<<<block_count, thread_count>>>(
        node_size_x, node_size_y, 
        flat_node2pin_start_map,
        flat_node2pin_map,
        node_ratios,
        num_nodes,
        pin_offset_x, pin_offset_y
        );
}

#define REGISTER_KERNEL_LAUNCHER(T)               \
    template void updatePinOffsetCudaLauncher<T>( \
        const T* node_size_x, const T* node_size_y, \
        const int *flat_node2pin_start_map,       \
        const int *flat_node2pin_map,             \
        const T *node_ratios,             \
        const int num_nodes,                      \
        T *pin_offset_x, T *pin_offset_y)

REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
