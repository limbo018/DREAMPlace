#include "hip/hip_runtime.h"
/**
 * @file   adjust_node_area_cuda_kernel.cu
 * @author Zixuan Jiang, Jiaqi Gu, Yibo Lin
 * @date   Dec 2019
 * @brief  Adjust cell area according to congestion map.
 */

#include "utility/src/utils.cuh"
// local dependency
#include "adjust_node_area/src/scaling_function.h"

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
__device__ DEFINE_AVERAGE_SCALING_FUNCTION(T); 

template <typename T>
__device__ DEFINE_MAX_SCALING_FUNCTION(T); 

template <typename T>
__global__ void computeInstanceRoutabilityOptimizationMap(
    const T *pos_x, const T *pos_y,
    const T *node_size_x, const T *node_size_y,
    const T *routing_utilization_map,
    T xl, T yl,
    T bin_size_x, T bin_size_y,
    int num_bins_x, int num_bins_y,
    int num_movable_nodes,
    T *instance_route_area)
{
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < num_movable_nodes)
    {
        const T x_min = pos_x[i];
        const T x_max = x_min + node_size_x[i];
        const T y_min = pos_y[i];
        const T y_max = y_min + node_size_y[i];

        // compute the bin box that this net will affect
        // We do NOT follow Wuxi's implementation. Instead, we clamp the bounding box.
        int bin_index_xl = int((x_min - xl) / bin_size_x);
        int bin_index_xh = int((x_max - xl) / bin_size_x) + 1;
        bin_index_xl = DREAMPLACE_STD_NAMESPACE::max(bin_index_xl, 0);
        bin_index_xh = DREAMPLACE_STD_NAMESPACE::min(bin_index_xh, num_bins_x);

        int bin_index_yl = int((y_min - yl) / bin_size_y);
        int bin_index_yh = int((y_max - yl) / bin_size_y) + 1;
        bin_index_yl = DREAMPLACE_STD_NAMESPACE::max(bin_index_yl, 0);
        bin_index_yh = DREAMPLACE_STD_NAMESPACE::min(bin_index_yh, num_bins_y);

        instance_route_area[i] = SCALING_OP(
                routing_utilization_map, 
                xl, yl, 
                bin_size_x, bin_size_y, 
                num_bins_x, num_bins_y, 
                bin_index_xl, 
                bin_index_yl, 
                bin_index_xh, 
                bin_index_yh, 
                x_min, y_min, x_max, y_max
                );
    }
}

template <typename T>
int computeInstanceRoutabilityOptimizationMapCudaLauncher(
    const T *pos_x, const T *pos_y,
    const T *node_size_x, const T *node_size_y,
    const T *routing_utilization_map,
    T xl, T yl,
    T bin_size_x, T bin_size_y,
    int num_bins_x, int num_bins_y,
    int num_movable_nodes,
    T *instance_route_area)
{
    int thread_count = 512;
    int block_count = ceilDiv(num_movable_nodes, thread_count);
    computeInstanceRoutabilityOptimizationMap<<<block_count, thread_count>>>(
        pos_x, pos_y,
        node_size_x, node_size_y,
        routing_utilization_map,
        xl, yl,
        bin_size_x, bin_size_y,
        num_bins_x, num_bins_y,
        num_movable_nodes,
        instance_route_area);
    return 0;
}

#define REGISTER_KERNEL_LAUNCHER(T)                                        \
    template int computeInstanceRoutabilityOptimizationMapCudaLauncher<T>( \
        const T *pos_x, const T *pos_y,                                    \
        const T *node_size_x, const T *node_size_y,                        \
        const T *routing_utilization_map,                                  \
        T xl, T yl,                                                        \
        T bin_size_x, T bin_size_y,                                        \
        int num_bins_x, int num_bins_y,                                    \
        int num_movable_nodes,                                             \
        T *instance_route_area)

REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
